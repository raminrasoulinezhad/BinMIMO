#include "hip/hip_runtime.h"
#include "GAMP_gpuFuncs.h" 
#include "myStructs.h"


__device__ void cuEstimX(hipComplex* out1, float* out2, float s1, float s2){
	float s2_inv;
	hipComplex s1_cpx;
	
	float logpxr1, logpxr2, logpxr3, logpxr4, max_log;
	float pxr1, pxr2, pxr3, pxr4, sum_pxr;
	float uvar1, uvar2, uvar3, uvar4;
	
	hipComplex x0_1(-0.7071, 0.7071);
	hipComplex x0_2(-0.7071, -0.7071);
	hipComplex x0_3(0.7071 , 0.7071);
	hipComplex x0_4(0.7071 , -0.7071);
	
	s2_inv = 1;///s2;
	s1_cpx.r = s1;
	logpxr1 = s2_inv*((s1_cpx - x0_1).magnitude2());
	logpxr2 = s2_inv*((s1_cpx - x0_2).magnitude2());
	logpxr3 = s2_inv*((s1_cpx - x0_3).magnitude2());
	logpxr4 = s2_inv*((s1_cpx - x0_4).magnitude2());
	
	max_log = logpxr1;
	if(logpxr2>max_log)
		max_log=logpxr2;
	if(logpxr3>max_log)
		max_log=logpxr3;
	if(logpxr4>max_log)
		max_log=logpxr4;
	
	logpxr1 = logpxr1 - max_log;
	logpxr2 = logpxr2 - max_log;
	logpxr3 = logpxr3 - max_log;
	logpxr4 = logpxr4 - max_log;
	
	pxr1 = exp(logpxr1);
	pxr2 = exp(logpxr2);
	pxr3 = exp(logpxr3);
	pxr4 = exp(logpxr4);
	
	sum_pxr = pxr1 + pxr2 + pxr3 + pxr4;
	
	pxr1 = pxr1;//sum_pxr;
	pxr2 = pxr2;//sum_pxr;
	pxr3 = pxr3;//sum_pxr;
	pxr4 = pxr4;//sum_pxr;
	
	*out1 = (hipComplex(pxr1,0)*x0_1 + hipComplex(pxr2,0)*x0_2 + hipComplex(pxr3,0)*x0_3 + hipComplex(pxr4,0)*x0_4);
	uvar1 = pxr1*((*out1-x0_1).magnitude2());
	uvar2 = pxr2*((*out1-x0_2).magnitude2());
	uvar3 = pxr3*((*out1-x0_3).magnitude2());
	uvar4 = pxr4*((*out1-x0_4).magnitude2());
	
	*out2 = (uvar1 + uvar2 + uvar3 + uvar4);
}

__device__ float cuMulReal(float* a, float* b, int row, int column, int di2, int di3){
	int k;
	float s = 0.0f;
	
	for (k=0; k<di2; k++){
		s += m(a,row,k,di2) * m(b,k,column,di3);
	}
	return s;
}
__device__ hipComplex cuMulCpx(hipComplex* a, hipComplex* b, int row, int column, int di2, int di3){
	int k;
	hipComplex s(0.0f, 0.0f);
	
	for (k=0; k<di2; k++){
		s = s + (m(a,row,k,di2) * m(b,k,column,di3));
	}
	return s;
}
__global__ void kernelFunc(float* a1d, float* b1d, hipComplex* c1d, float* a2d, float* b2d, hipComplex* c2d, int di1, int di2, int di3) {

	int tx = threadIdx.x;
	int by = blockIdx.y;
	int bx = blockIdx.x;
	int row, column;
	float s1 = 0.0f;
	float s2 = 0.0f;
	hipComplex tmp(0);
	hipComplex* out1=&tmp;
	float tmp2=0.0f;
	float* out2=&tmp2;
	
	row = bx;
	column = (by)*(blockDim.x)+tx;
	
	s1 = cuMulReal(a1d, b1d, row, column, di2, di3);
	
	s2 = cuMulReal(a2d, b2d, row, column, di2, di3);
	
	cuEstimX(out1, out2, s1, s2);
	
	m(c1d,row,column,di3) = hipComplex(*out1);
	m(c2d,row,column,di3) = hipComplex(*out2);
}

__global__ void pCalc(hipComplex* hhat, hipComplex* xhat, float* vx, float* vh, hipComplex* shat, float* vp, hipComplex* phat, int N, int K, int data_len, int pilot_len)
{
	int tx = threadIdx.x;
	int by = blockIdx.y;
	int bx = blockIdx.x;
	int row, column;
	float vpbar = 0.0f;
	float vptemp = 0.0f;
	hipComplex pbar(0,0);
	float vhi, vxi;
	hipComplex hhati, xhati;
	
	const int T = data_len + pilot_len;
	row = bx;
	column = (by)*(blockDim.x)+tx;
	
	for(int i=0; i<K; i++)
	{
		vhi = m(vh,row,i,K);
		vxi = m(vx,i,column,T);
		hhati = m(hhat,row,i,K);
		xhati = m(xhat,i,column,T);
		
		vpbar += (hhati.magnitude2())*vxi + vhi*(xhati.magnitude2());
		pbar = pbar + (hhati*xhati);
		vptemp += vhi*vxi; 
	}
	vpThread = vpbar+vptemp;
	phatThread = pbar-(m(shat,row,column,T)*((hipComplex)vpbar));
	
	m(vp,row,column,T) = vpThread;
	m(phat,row,column,T) = phatThread;
}
//-----------------------------------------------------------------------------
void gpuKernel(hipComplex* hhat, hipComplex* xhat, float* vx, float* vh, hipComplex* shat, float* vp, hipComplex* phat, int N, int K, int data_len, int pilot_len, double* gpu_kernel_time) {
	// allocate memory on GPU
	// copy data to GPU
	// call kernelFunc
	// copy the results back to CPU
	// free GPU memory
	float *vxd, *vhd, *vpd;
	hipComplex *hhatd, *xhatd, *shatd, *phat;
	
	HANDLE_ERROR(hipMalloc((void**)&vxd, K*T*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&vhd, N*K*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&vpd, N*T*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&shatd, N*T*sizeof(hipComplex)));
	HANDLE_ERROR(hipMalloc((void**)&xhatd, K*T*sizeof(hipComplex)));
	HANDLE_ERROR(hipMalloc((void**)&hhatd, N*K*sizeof(hipComplex)));
	HANDLE_ERROR(hipMalloc((void**)&phatd, N*T*sizeof(hipComplex)));
	
	HANDLE_ERROR(hipMemcpy(vxd, vx, K*T*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(vhd, vh, N*K*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(hhatd, hhat, N*K*sizeof(hipComplex), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(xhatd, xhat, K*T*sizeof(hipComplex), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(shatd, shat, N*T*sizeof(hipComplex), hipMemcpyHostToDevice));
	
	GpuTimer timer;
    timer.Start();
	pCalc<<< dim3(N,1,1), T >>>(hipComplex* hhatd, hipComplex* xhatd, float* vxd, float* vhd, hipComplex shatd, float* vpd, hipComplex* phatd, int N, int K, int data_len, int pilot_len)
	timer.Stop();
	*gpu_kernel_time = timer.Elapsed();
	
	HANDLE_ERROR(hipMemcpy(phat, phatd, N*T*sizeof(hipComplex), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(vhd, vh, N*K*sizeof(float), hipMemcpyHostToDevice));
	
	HANDLE_ERROR(hipFree(vx));
	HANDLE_ERROR(hipFree(vh));
	HANDLE_ERROR(hipFree(vp));
	HANDLE_ERROR(hipFree(shat));
	HANDLE_ERROR(hipFree(xhat));
	HANDLE_ERROR(hipFree(hhat));
	HANDLE_ERROR(hipFree(phat));
}
void gpuKernelOld(float* a1, float* b1, hipComplex* c1, float* a2, float* b2, hipComplex* c2, int di1, int di2, int di3, double* gpu_kernel_time) {
	// allocate memory on GPU
	// copy data to GPU
	// call kernelFunc
	// copy the results back to CPU
	// free GPU memory
	float *a1d, *b1d;
	float *a2d, *b2d;
	hipComplex *c1d, *c2d;
	
	HANDLE_ERROR(hipMalloc((void**)&a1d, di1*di2*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b1d, di2*di3*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&c1d, di1*di3*sizeof(hipComplex)));
	HANDLE_ERROR(hipMalloc((void**)&a2d, di1*di2*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b2d, di2*di3*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&c2d, di1*di3*sizeof(hipComplex)));
	
	HANDLE_ERROR(hipMemcpy(a1d, a1, di1*di2*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b1d, b1, di2*di3*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a2d, a2, di1*di2*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b2d, b2, di2*di3*sizeof(float), hipMemcpyHostToDevice));
	
	GpuTimer timer;
    timer.Start();
	kernelFunc<<< dim3(di1,1,1), di3 >>>(a1d, b1d, c1d, a2d, b2d, c2d, di1, di2, di3);
	timer.Stop();
	*gpu_kernel_time = timer.Elapsed();
	
	HANDLE_ERROR(hipMemcpy(c1, c1d, di1*di3*sizeof(hipComplex), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(c2, c2d, di1*di3*sizeof(hipComplex), hipMemcpyDeviceToHost));
	
	HANDLE_ERROR(hipFree(a1d));
	HANDLE_ERROR(hipFree(b1d));
	HANDLE_ERROR(hipFree(c1d));
	HANDLE_ERROR(hipFree(a2d));
	HANDLE_ERROR(hipFree(b2d));
	HANDLE_ERROR(hipFree(c2d));
}