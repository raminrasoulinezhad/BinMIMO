#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define m(data,y,x,dim)		data[y*dim+x]
#define MAX_THREADS		1024

struct hipComplex {
	float r;
	float i;
	__device__ hipComplex() : r(0), i(0) {}
	__device__ hipComplex( float a) : r(a), i(0) {}
	__device__ hipComplex( float a, float b ) : r(a), i(b) {}
	float real(void){return r;}
	float imag(void){return i;}
	__device__ float magnitude2( void ) {
	return r * r + i * i;
	}
	__device__ hipComplex operator*(const hipComplex& a) {
	return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	__device__ hipComplex operator+(const hipComplex& a) {
	return hipComplex(r+a.r, i+a.i);
	}
	__device__ hipComplex operator-(const hipComplex& a) {
	return hipComplex(r-a.r, i-a.i);
	}
};
struct myComplex {
	float r;
	float i;
	myComplex() : r(0), i(0) {}
	myComplex( float a) : r(a), i(0) {}
	myComplex( float a, float b ) : r(a), i(b) {}
	float real(void){return r;}
	float imag(void){return i;}
	float magnitude2( void ) {
	return r * r + i * i;
	}
	myComplex operator*(const myComplex& a) {
	return myComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	myComplex operator+(const myComplex& a) {
	return myComplex(r+a.r, i+a.i);
	}
	myComplex operator-(const myComplex& a) {
	return myComplex(r-a.r, i-a.i);
	}
	myComplex operator/(const hipComplex& a) {
	return myComplex(r-a.r, i-a.i);
	}
};

// ===========================> Functions Prototype <===============================
void fill(float* data, int size);
float maxi(float in1, float in2);
void estimX(myComplex v, myComplex wvar, myComplex* umean, myComplex* uvar);
double calc_mse(float* data1, float* data2, int size);
double calc_mse(myComplex* data1, hipComplex* data2, int size);
void cpuMul_yx(float* a, float* b, myComplex* c, int di2, int di3, int y, int x);
void cpuMul(float* a, float* b, myComplex* c, int di1, int di2, int di3);
void cpuKernel(float* a1, float* b1, myComplex* c1, float* a2, float* b2, myComplex* c2, int di1, int di2, int di3);
__device__ float cuMulReal(float* a, float* b, int row, int column, int di2, int di3);
__device__ void cuEstimX(hipComplex* out1, float* out2, float s1, float s2);
__global__ void kernelFunc(float* a1d, float* b1d, hipComplex* c1d, float* a2d, float* b2d, hipComplex* c2d, int di1, int di2, int di3);
void gpuKernel(float* a1, float* b1, hipComplex* c1, float* a2, float* b2, hipComplex* c2, int di1, int di2, int di3);
// =================================================================================

int main(int argc, char** argv) {

    struct hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);
    printf("Device Name: %s\n", p.name);
	printf("shared mem size: %d\n", p.regsPerBlock);
	
	// get parameter from command line to build Matrix dimension
	const int di1 = 200;
	const int di2 = 64;
	const int di3 = 450;
	
	// allocate memory in CPU for calculation
	float* a1;
	float* b1;
	myComplex* c1_serial;
	hipComplex* c1;
	float* a2;
	float* b2;
	myComplex* c2_serial;
	hipComplex* c2;
	a1        = (float*)malloc(di1*di2 * sizeof(float));
	b1        = (float*)malloc(di2*di3 * sizeof(float));
	c1_serial = (myComplex*)malloc(di1*di3 * sizeof(myComplex));
	c1        = (hipComplex*)malloc(di1*di3 * sizeof(hipComplex));
	a2        = (float*)malloc(di1*di2 * sizeof(float));
	b2        = (float*)malloc(di2*di3 * sizeof(float));
	c2_serial = (myComplex*)malloc(di1*di3 * sizeof(myComplex));
	c2        = (hipComplex*)malloc(di1*di3 * sizeof(hipComplex));
		
	// fill a, b matrices with random values between -16.0f and 16.0f
	srand(0);
	fill(a1, di1*di2);
	fill(b1, di2*di3);
	fill(a2, di1*di2);
	fill(b2, di2*di3);
	
	// time measurement for CPU calculation
	clock_t t0 = clock();
	cpuKernel (a1, b1, c1_serial, a2, b2, c2_serial, di1, di2, di3);
	clock_t t1 = clock();
		
	// time measurement for GPU calculation
	clock_t t2 = clock();
	gpuKernel (a1, b1, c1, a2, b2, c2, di1, di2, di3);
	clock_t t3 = clock();

	// check correctness of calculation
	float mse;
	mse = calc_mse( c1_serial, c1, di1*di3 ) + calc_mse( c2_serial, c2, di1*di3 );

	printf("dim1=%d dim2=%d dim3=%d\t CPU=%06ld ms GPU=%06ld ms mse=%f\n",di1, di2, di3, (t1-t0)/1000, (t3-t2)/1000, mse);
		
	// free allocated memory for later use
	free(a1);
	free(b1);
	free(c1_serial);
	free(c1);
	free(a2);
	free(b2);
	free(c2_serial);
	free(c2);
   
	return 0;
}

//-----------------------------------------------------------------------------
void fill(float* data, int size) {
    for (int i=0; i<size; ++i)
        data[i] = (float) (rand() % 33 - 16);
}

double calc_mse (myComplex* data1, hipComplex* data2, int size) {

	double mse = 0.0;
	int i; for (i=0; i<size; i++) {
		myComplex diff = (data1[i]/data2[i]);
		float e = diff.magnitude2();
		// printf("1r=%.4f, 1i=%.2f, 2r=%.4f, 2i=%.2f",data1[i].real(), data1[i].imag(), data2[i].real(), data2[i].imag());
		// printf("  diffR=%.2f, diffI=%.2f, e=%f\r\n",diff.real(), diff.imag(),e);
		mse += e;
		// printf("i=%d, mse=%f ",i,mse);
	}
	// i=1839;
	// printf("1r=%.4f, 1i=%.2f, 2r=%.4f, 2i=%.2f\r\n",data1[i].real(), data1[i].imag(), data2[i].real(), data2[i].imag());
	mse = mse / size;
	return mse;
}

double calc_mse (float* data1, float* data2, int size) {

	double mse = 0.0;
	int i; for (i=0; i<size; i++) {
		double e = data1[i]-data2[i];
		e = e * e;
		mse += e;
	}
	mse = mse / size;
	return mse;
}

//-----------------------------------------------------------------------------
void cpuMul_yx(float* a, float* b, myComplex* c, int di2, int di3, int y, int x) { // one element
	m(c,y,x,di3)=0;
    for(int k=0; k<di2; k++) {
		m(c,y,x,di3) = m(c,y,x,di3) + myComplex(m(a,y,k,di2) * m(b,k,x,di3));
	}
}
void cpuMul(float* a, float* b, myComplex* c, int di1, int di2, int di3) { // entire matrix
    for(int y=0; y<di1; y++)
    for(int x=0; x<di3; x++)
	{
		cpuMul_yx(a,b,c,di2,di3,y,x);
	}
}
void cpuKernel(float* a1, float* b1, myComplex* c1, float* a2, float* b2, myComplex* c2, int di1, int di2, int di3) { // entire matrix
	int i=0;
	// myComplex umean;
	// float uvar;
	
	cpuMul(a1,b1,c1,di1,di2,di3);
	cpuMul(a2,b2,c2,di1,di2,di3);
	for(i=0; i<di1*di3; i++)
		estimX(c1[i], c2[i], c1+i, c2+i);
	// printf("umean=%f, uvar=%f", creal(umean), creal(uvar));
}

float maxi(float in1, float in2) {
	if (in1 > in2)
		return in1;
	else
		return in2;
}

void estimX(myComplex v, myComplex wvar, myComplex* umean, myComplex* uvar) {

	float wvar_inv;
	float logpxr1, logpxr2, logpxr3, logpxr4, max_log;
	float pxr1, pxr2, pxr3, pxr4, sum_pxr;
	float uvar1, uvar2, uvar3, uvar4;
	
	myComplex x0_1(-0.7071, 0.7071);
	myComplex x0_2(-0.7071, -0.7071);
	myComplex x0_3(0.7071, 0.7071);
	myComplex x0_4(0.7071, -0.7071);
	
	wvar_inv = 1;///(wvar.r);//imag=0 ast
	logpxr1 = wvar_inv*((v - x0_1).magnitude2());
	logpxr2 = wvar_inv*((v - x0_2).magnitude2());
	logpxr3 = wvar_inv*((v - x0_3).magnitude2());
	logpxr4 = wvar_inv*((v - x0_4).magnitude2());
	
	max_log = maxi(maxi(logpxr1,logpxr2),maxi(logpxr3,logpxr4));
	
	logpxr1 = logpxr1 - max_log;
	logpxr2 = logpxr2 - max_log;
	logpxr3 = logpxr3 - max_log;
	logpxr4 = logpxr4 - max_log;
	
	pxr1 = exp(logpxr1);
	pxr2 = exp(logpxr2);
	pxr3 = exp(logpxr3);
	pxr4 = exp(logpxr4);
	
	sum_pxr = pxr1 + pxr2 + pxr3 + pxr4;
	
	pxr1 = pxr1;///sum_pxr;
	pxr2 = pxr2;///sum_pxr;
	pxr3 = pxr3;///sum_pxr;
	pxr4 = pxr4;///sum_pxr;
	
	*umean = myComplex(pxr1,0)*x0_1 + myComplex(pxr2,0)*x0_2 + myComplex(pxr3,0)*x0_3 + myComplex(pxr4,0)*x0_4;
	// printf("umeanR=%.3f, I=%.3f\r\n", (*umean-x0_1).r, (*umean-x0_1).i);
	uvar1 = pxr1*((*umean-x0_1).magnitude2());
	uvar2 = pxr2*((*umean-x0_2).magnitude2());
	uvar3 = pxr3*((*umean-x0_3).magnitude2());
	uvar4 = pxr4*((*umean-x0_4).magnitude2());
	
	*uvar = myComplex(uvar1 + uvar2 + uvar3 + uvar4);
	
}

__device__ void cuEstimX(hipComplex* out1, float* out2, float s1, float s2){
	float s2_inv;
	hipComplex s1_cpx;
	
	float logpxr1, logpxr2, logpxr3, logpxr4, max_log;
	float pxr1, pxr2, pxr3, pxr4, sum_pxr;
	float uvar1, uvar2, uvar3, uvar4;
	
	hipComplex x0_1(-0.7071, 0.7071);
	hipComplex x0_2(-0.7071, -0.7071);
	hipComplex x0_3(0.7071 , 0.7071);
	hipComplex x0_4(0.7071 , -0.7071);
	
	s2_inv = 1;///s2;
	s1_cpx.r = s1;
	logpxr1 = s2_inv*((s1_cpx - x0_1).magnitude2());
	logpxr2 = s2_inv*((s1_cpx - x0_2).magnitude2());
	logpxr3 = s2_inv*((s1_cpx - x0_3).magnitude2());
	logpxr4 = s2_inv*((s1_cpx - x0_4).magnitude2());
	
	max_log = logpxr1;
	if(logpxr2>max_log)
		max_log=logpxr2;
	if(logpxr3>max_log)
		max_log=logpxr3;
	if(logpxr4>max_log)
		max_log=logpxr4;
	
	logpxr1 = logpxr1 - max_log;
	logpxr2 = logpxr2 - max_log;
	logpxr3 = logpxr3 - max_log;
	logpxr4 = logpxr4 - max_log;
	
	pxr1 = exp(logpxr1);
	pxr2 = exp(logpxr2);
	pxr3 = exp(logpxr3);
	pxr4 = exp(logpxr4);
	
	sum_pxr = pxr1 + pxr2 + pxr3 + pxr4;
	
	pxr1 = pxr1;//sum_pxr;
	pxr2 = pxr2;//sum_pxr;
	pxr3 = pxr3;//sum_pxr;
	pxr4 = pxr4;//sum_pxr;
	
	*out1 = (hipComplex(pxr1,0)*x0_1 + hipComplex(pxr2,0)*x0_2 + hipComplex(pxr3,0)*x0_3 + hipComplex(pxr4,0)*x0_4);
	uvar1 = pxr1*((*out1-x0_1).magnitude2());
	uvar2 = pxr2*((*out1-x0_2).magnitude2());
	uvar3 = pxr3*((*out1-x0_3).magnitude2());
	uvar4 = pxr4*((*out1-x0_4).magnitude2());
	
	*out2 = (uvar1 + uvar2 + uvar3 + uvar4);
}

__device__ float cuMulReal(float* a, float* b, int row, int column, int di2, int di3){
	int k;
	float s = 0.0f;
	
	for (k=0; k<di2; k++){
		s += m(a,row,k,di2) * m(b,k,column,di3);
	}
	return s;
}
__device__ hipComplex cuMulCpx(hipComplex* a, hipComplex* b, int row, int column, int di2, int di3){
	int k;
	hipComplex s(0.0f, 0.0f);
	
	for (k=0; k<di2; k++){
		s = s + (m(a,row,k,di2) * m(b,k,column,di3));
	}
	return s;
}
__global__ void kernelFunc(float* a1d, float* b1d, hipComplex* c1d, float* a2d, float* b2d, hipComplex* c2d, int di1, int di2, int di3) {

	int tx = threadIdx.x;
	int by = blockIdx.y;
	int bx = blockIdx.x;
	int row, column;
	float s1 = 0.0f;
	float s2 = 0.0f;
	hipComplex tmp(0);
	hipComplex* out1=&tmp;
	float tmp2=0.0f;
	float* out2=&tmp2;
	
	row = bx;
	column = (by)*(blockDim.x)+tx;
	
	s1 = cuMulReal(a1d, b1d, row, column, di2, di3);
	
	s2 = cuMulReal(a2d, b2d, row, column, di2, di3);
	
	cuEstimX(out1, out2, s1, s2);
	
	m(c1d,row,column,di3) = hipComplex(*out1);
	m(c2d,row,column,di3) = hipComplex(*out2);
}
//-----------------------------------------------------------------------------
void gpuKernel(float* a1, float* b1, hipComplex* c1, float* a2, float* b2, hipComplex* c2, int di1, int di2, int di3) {
	// allocate memory on GPU
	// copy data to GPU
	// call kernelFunc
	// copy the results back to CPU
	// free GPU memory
	float *a1d, *b1d;
	float *a2d, *b2d;
	hipComplex *c1d, *c2d;
	
	hipMalloc((void**)&a1d, di1*di2*sizeof(float));
	hipMalloc((void**)&b1d, di2*di3*sizeof(float));
	hipMalloc((void**)&c1d, di1*di3*sizeof(hipComplex));
	hipMalloc((void**)&a2d, di1*di2*sizeof(float));
	hipMalloc((void**)&b2d, di2*di3*sizeof(float));
	hipMalloc((void**)&c2d, di1*di3*sizeof(hipComplex));
	
	hipMemcpy(a1d, a1, di1*di2*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b1d, b1, di2*di3*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(a2d, a2, di1*di2*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b2d, b2, di2*di3*sizeof(float), hipMemcpyHostToDevice);
	
	//kernelFunc <<<dim3(n/MAX_THREADS,n/MAX_THREADS,MAX_THREADS),MAX_THREADS>>> (ad,bd,cd,n);
	kernelFunc<<< dim3(di1,1,1), di3 >>>(a1d, b1d, c1d, a2d, b2d, c2d, di1, di2, di3);
	
	hipMemcpy(c1, c1d, di1*di3*sizeof(hipComplex), hipMemcpyDeviceToHost);
	hipMemcpy(c2, c2d, di1*di3*sizeof(hipComplex), hipMemcpyDeviceToHost);
	
	hipFree(a1d);
	hipFree(b1d);
	hipFree(c1d);
	hipFree(a2d);
	hipFree(b2d);
	hipFree(c2d);
}